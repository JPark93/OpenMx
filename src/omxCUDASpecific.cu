#include "hip/hip_runtime.h"
/*
 *  Copyright 2021 by the individuals mentioned in the source code history
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 */
#include "omxCUDASpecific.h"
#include <stdio.h>

// Squares each cell value in an array of integers
__global__ void square_array(int *array, int arrayCount)
{
 int idx = threadIdx.x + blockIdx.x * blockDim.x;
 if (idx < arrayCount)
 {
   array[idx] *= array[idx];
 }
}

void callHelloWorld(){
  printf("Hello, World from .cu!\n");

  int n = 1024;

  int *array;
  size_t bytes = n*sizeof(int);
  array = (int*)malloc(bytes);

  for (int i =0; i <n; i++){
    array[i] = i;
  }

  int *d_a;

  hipMalloc(&d_a, n*sizeof(int));

  int blockSize;
  int minGridSize;
  int gridSize;

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
                                      square_array);
  // Round up according to array size
  gridSize = (n + blockSize - 1) / blockSize;

  hipMemcpy(d_a, array, n*sizeof(int), hipMemcpyHostToDevice);

  square_array<<< gridSize, blockSize >>>(d_a, n);

  hipDeviceSynchronize();

  hipMemcpy(array, d_a, n*sizeof(int), hipMemcpyDeviceToHost);

  for (int i =0; i <n; i+=100){
    printf("Square of %d is %d\n", i, array[i]);
  }
}
