/*
 *  Copyright 2021 by the individuals mentioned in the source code history
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 */
#include <iostream>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include <hipsolver.h>

#include "omxCUDASpecific.h"

void gpuCholeskyInvertAndDiag(double* h_input, double* h_result, double* h_diag, int N, int* h_devinfo)
{
  // Instantiate solver GPU context
  hipsolverHandle_t solver_handle;
  hipsolverDnCreate(&solver_handle);

  // Allocate memory and copy input array to GPU
  double *d_input;
  hipMalloc(&d_input, N*N*sizeof(double));
  hipMemcpy(d_input, h_input, N*N*sizeof(double), hipMemcpyHostToDevice);

  // Determine block allocations for optimized Cholesky factorization
  int worksize = 0;
  hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_UPPER, N, d_input, N, &worksize);
  double *work;
  hipMalloc(&work, worksize*sizeof(double));
  int* d_devinfo;
  hipMalloc(&d_devinfo, sizeof(int));

  // Run Cholesky Factorization
  hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, N, d_input, N, work, worksize, d_devinfo);
  hipMemcpy(h_devinfo, d_devinfo, sizeof(int), hipMemcpyDeviceToHost);
  //std::cout << *h_devinfo <<std::endl;
  if (*h_devinfo != 0){
    std::cout << "GPU Cholesky Factorization failed!" << std::endl;
    return;
  }
  getDiagonalFromDevice(d_input, h_diag, N);

  // Setup Identity Matrix & Solve For Inversion
  // Inversion result modifies d_identity in place
  double *h_identity = (double*)malloc(N*N*sizeof(double));
  initIdentityMatrix(h_identity, N);
  double *d_identity;
  hipMalloc(&d_identity, N*N*sizeof(double));
  hipMemcpy(d_identity, h_identity, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipsolverDnDpotrs(solver_handle, HIPBLAS_FILL_MODE_UPPER, N, N, d_input, N, d_identity, N, d_devinfo);
  hipMemcpy(h_devinfo, d_devinfo, sizeof(int), hipMemcpyDeviceToHost);
  if (*h_devinfo != 0){
    std::cout << "GPU Solve for Inversion failed!" << std::endl;
    return;
  }
  hipMemcpy(h_result, d_identity, N*N*sizeof(double), hipMemcpyDeviceToHost);
  return;
}

void initIdentityMatrix(double* array, int N){
    for (int i=0; i < N*N; i++)
    {
    	if ((i-i/N)%N == 0) array[i] = 1.0;
    	else array[i] = 0.0;
    }
    return;
}

void getDiagonalFromDevice(double* d_array, double* h_diagvec, int N){
	for(int i = 0; i < N; i++){
		hipMemcpy(&h_diagvec[i], &d_array[i+i*N], sizeof(double), hipMemcpyDeviceToHost);
	}
  return;
}
